#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"

#include "tensor.hu"
#include "utils.h"

__global__ void tensor_add_kernel(const float *A, const float *B, float *out, size_t elems_this_chunk) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < elems_this_chunk)
        out[idx] = A[idx] + B[idx];
}

__global__ void tensor_sub_kernel(const float *A, const float *B, float *out, size_t elems_this_chunk) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < elems_this_chunk)
        out[idx] = A[idx] - B[idx];
}

__global__ void tensor_scale_kernel(const float *A, const float alpha, float *out, size_t elems_this_chunk) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < elems_this_chunk)
        out[idx] = A[idx] * alpha;
}

__global__ void tensor_mul_kernel(const float *A, const float *B, float *out, size_t elems_this_chunk) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < elems_this_chunk)
        out[idx] = A[idx] * B[idx];
}

__global__ void tensor_ones_kernel(float *data, size_t elems_this_chunk) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < elems_this_chunk)
        data[idx] = 1.0f;
}

__global__ void tensor_zeros_kernel(float *data, size_t elems_this_chunk) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < elems_this_chunk)
        data[idx] = 0.0f;
}

__global__ void tensor_rand_kernel(float *data, size_t elems_this_chunk, unsigned long long seed) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < elems_this_chunk) {
        hiprandState_t state;
        hiprand_init(seed, idx, 0, &state);

        data[idx] = hiprand_uniform(&state);
    }
}

__global__ void matmul_chunk_kernel( const float *A_sub, const float *B_sub, float *C_sub, int m, int p, int n_sub) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < m && col < p) {
        float sum = 0.0f;
        for (int k = 0; k < n_sub; ++k)
            sum += A_sub[row * n_sub + k] * B_sub[k * p + col];
        C_sub[row * p + col] += sum;
    }
}

Tensor *tensor_new(int ndim, const int *shape) {
    Tensor *t = (Tensor*)malloc(sizeof(Tensor));
    if (!t) {
        fprintf(stderr, "Error allocating Tensor\n");
        return NULL;
    }

    t->ndim = ndim;

    t->shape = (int*)malloc(ndim * sizeof(int));
    if (!t->shape) {
        fprintf(stderr, "Error allocating shape\n");
        free(t);
        return NULL;
    }
    memcpy(t->shape, shape, ndim * sizeof(int));

    size_t size = 1;
    for (int i = 0; i < ndim; i++) {
        size *= shape[i];
    }
    t->size = size;

    t->stride = (int*)malloc(ndim * sizeof(int));
    if (!t->stride) {
        fprintf(stderr, "Error allocating stride\n");
        free(t->shape);
        free(t);
        return NULL;
    }

    t->stride[ndim-1] = 1;
    for (int i = ndim-2; i >= 0; i--) {
        t->stride[i] = t->stride[i+1] * shape[i+1];
    }

    t->data = (float*)malloc(size * sizeof(float));
    if (!t->data) {
        fprintf(stderr, "Error allocating data\n");
        free(t->stride);
        free(t->shape);
        free(t);
        return NULL;
    }

    memset(t->data, 0, size * sizeof(float));

    return t;
}

void tensor_add_cuda(Tensor *out, Tensor *A, Tensor *B, size_t chunk_size) {
    if (A->ndim != B->ndim || A->ndim != out->ndim) {
        fprintf(stderr, "tensor_add_cuda: incompatible dimensions (ndim)\n");
        exit(EXIT_FAILURE);
    }
    for (int i = 0; i < A->ndim; ++i) {
        if (A->shape[i] != B->shape[i] || A->shape[i] != out->shape[i]) {
            fprintf(stderr, "tensor_add_cuda: diferent shapes in dimension %d (A=%d, B=%d, out=%d)\n", i, A->shape[i], B->shape[i], out->shape[i]);
            exit(EXIT_FAILURE);
        }
    }

    size_t N = A->size;
    size_t num_chunks = (N + chunk_size - 1) / chunk_size;

    size_t bytes_chunk = chunk_size * sizeof(float);
    float *d_A = NULL, *d_B = NULL, *d_out = NULL;
    
    hipMalloc((void**)&d_A, bytes_chunk);
    hipMalloc((void**)&d_B, bytes_chunk);
    hipMalloc((void**)&d_out, bytes_chunk);

    const int threads_per_block = 256;

    for (size_t chunk_idx = 0; chunk_idx < num_chunks; ++chunk_idx) {
        size_t offset = chunk_idx * chunk_size;

        size_t elems_this_chunk = chunk_size;
        if (offset + elems_this_chunk > N) {
            elems_this_chunk = N - offset;
        }
        size_t bytes_this_chunk = elems_this_chunk * sizeof(float);

        hipMemcpy(d_A, A->data + offset, bytes_this_chunk, hipMemcpyHostToDevice);
        hipMemcpy(d_B, B->data + offset, bytes_this_chunk, hipMemcpyHostToDevice);

        int blocks_per_grid = (int)((elems_this_chunk + threads_per_block - 1) / threads_per_block);
        tensor_add_kernel<<<blocks_per_grid, threads_per_block>>>(d_A, d_B, d_out, elems_this_chunk);

        hipGetLastError();
        hipDeviceSynchronize();

        hipMemcpy(out->data + offset, d_out, bytes_this_chunk, hipMemcpyDeviceToHost);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_out);
}

void tensor_sub_cuda(Tensor *out, Tensor *A, Tensor *B, size_t chunk_size) {
    if (A->ndim != B->ndim || A->ndim != out->ndim) {
        fprintf(stderr, "tensor_sub_cuda: incompatible dimensions (ndim)\n");
        exit(EXIT_FAILURE);
    }
    for (int i = 0; i < A->ndim; ++i) {
        if (A->shape[i] != B->shape[i] || A->shape[i] != out->shape[i]) {
            fprintf(stderr, "tensor_sub_cuda: diferent shapes in dimension %d (A=%d, B=%d, out=%d)\n", i, A->shape[i], B->shape[i], out->shape[i]);
            exit(EXIT_FAILURE);
        }
    }

    size_t N = A->size;
    size_t num_chunks = (N + chunk_size - 1) / chunk_size;

    size_t bytes_chunk = chunk_size * sizeof(float);
    float *d_A = NULL, *d_B = NULL, *d_out = NULL;
    
    hipMalloc((void**)&d_A, bytes_chunk);
    hipMalloc((void**)&d_B, bytes_chunk);
    hipMalloc((void**)&d_out, bytes_chunk);

    const int threads_per_block = 256;

    for (size_t chunk_idx = 0; chunk_idx < num_chunks; ++chunk_idx) {
        size_t offset = chunk_idx * chunk_size;

        size_t elems_this_chunk = chunk_size;
        if (offset + elems_this_chunk > N) {
            elems_this_chunk = N - offset;
        }
        size_t bytes_this_chunk = elems_this_chunk * sizeof(float);

        hipMemcpy(d_A, A->data + offset, bytes_this_chunk, hipMemcpyHostToDevice);
        hipMemcpy(d_B, B->data + offset, bytes_this_chunk, hipMemcpyHostToDevice);

        int blocks_per_grid = (int)((elems_this_chunk + threads_per_block - 1) / threads_per_block);
        tensor_sub_kernel<<<blocks_per_grid, threads_per_block>>>(d_A, d_B, d_out, elems_this_chunk);

        hipGetLastError();
        hipDeviceSynchronize();

        hipMemcpy(out->data + offset, d_out, bytes_this_chunk, hipMemcpyDeviceToHost);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_out);
}

void tensor_scale_cuda(Tensor *out, Tensor *A, float alpha, size_t chunk_size) {
    if (A->ndim != out->ndim) {
        fprintf(stderr, "tensor_scale_cuda: incompatible dimensions (ndim)\n");
        exit(EXIT_FAILURE);
    }
    for (int i = 0; i < A->ndim; ++i) {
        if (A->shape[i] != out->shape[i]) {
            fprintf(stderr, "tensor_scale_cuda: diferent shapes in dimension %d (A=%d, out=%d)\n", i, A->shape[i], out->shape[i]);
            exit(EXIT_FAILURE);
        }
    }

    size_t N = A->size;
    size_t num_chunks = (N + chunk_size - 1) / chunk_size;

    size_t bytes_chunk = chunk_size * sizeof(float);
    float *d_A = NULL, *d_out = NULL;
    
    hipMalloc((void**)&d_A, bytes_chunk);
    hipMalloc((void**)&d_out, bytes_chunk);

    const int threads_per_block = 256;

    for (size_t chunk_idx = 0; chunk_idx < num_chunks; ++chunk_idx) {
        size_t offset = chunk_idx * chunk_size;

        size_t elems_this_chunk = chunk_size;
        if (offset + elems_this_chunk > N) {
            elems_this_chunk = N - offset;
        }
        size_t bytes_this_chunk = elems_this_chunk * sizeof(float);

        hipMemcpy(d_A, A->data + offset, bytes_this_chunk, hipMemcpyHostToDevice);

        int blocks_per_grid = (int)((elems_this_chunk + threads_per_block - 1) / threads_per_block);
        tensor_scale_kernel<<<blocks_per_grid, threads_per_block>>>(d_A, alpha, d_out, elems_this_chunk);

        hipGetLastError();
        hipDeviceSynchronize();

        hipMemcpy(out->data + offset, d_out, bytes_this_chunk, hipMemcpyDeviceToHost);
    }

    hipFree(d_A);
    hipFree(d_out);
}

void tensor_mul_cuda(Tensor *out, Tensor *A, Tensor *B, size_t chunk_size) {
    if (A->ndim != B->ndim || A->ndim != out->ndim) {
        fprintf(stderr, "tensor_mul_cuda: incompatible dimensions (ndim)\n");
        exit(EXIT_FAILURE);
    }
    for (int i = 0; i < A->ndim; ++i) {
        if (A->shape[i] != B->shape[i] || A->shape[i] != out->shape[i]) {
            fprintf(stderr, "tensor_mul_cuda: diferent shapes in dimension %d (A=%d, B=%d, out=%d)\n", i, A->shape[i], B->shape[i], out->shape[i]);
            exit(EXIT_FAILURE);
        }
    }

    size_t N = A->size;
    size_t num_chunks = (N + chunk_size - 1) / chunk_size;

    size_t bytes_chunk = chunk_size * sizeof(float);
    float *d_A = NULL, *d_B = NULL, *d_out = NULL;
    
    hipMalloc((void**)&d_A, bytes_chunk);
    hipMalloc((void**)&d_B, bytes_chunk);
    hipMalloc((void**)&d_out, bytes_chunk);

    const int threads_per_block = 256;

    for (size_t chunk_idx = 0; chunk_idx < num_chunks; ++chunk_idx) {
        size_t offset = chunk_idx * chunk_size;

        size_t elems_this_chunk = chunk_size;
        if (offset + elems_this_chunk > N) {
            elems_this_chunk = N - offset;
        }
        size_t bytes_this_chunk = elems_this_chunk * sizeof(float);

        hipMemcpy(d_A, A->data + offset, bytes_this_chunk, hipMemcpyHostToDevice);
        hipMemcpy(d_B, B->data + offset, bytes_this_chunk, hipMemcpyHostToDevice);

        int blocks_per_grid = (int)((elems_this_chunk + threads_per_block - 1) / threads_per_block);
        tensor_mul_kernel<<<blocks_per_grid, threads_per_block>>>(d_A, d_B, d_out, elems_this_chunk);

        hipGetLastError();
        hipDeviceSynchronize();

        hipMemcpy(out->data + offset, d_out, bytes_this_chunk, hipMemcpyDeviceToHost);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_out);
}

void tensor_matmul_cuda(Tensor *out, const Tensor *A, const Tensor *B, size_t chunk_size) {
    if (A->ndim != 2 || B->ndim != 2) {
        fprintf(stderr, "tensor_matmul_cuda_chunked: only 2d tensors are suported (A.ndim=%d, B.ndim=%d)\n", A->ndim, B->ndim);
        exit(EXIT_FAILURE);
    }
    int m = A->shape[0];
    int n = A->shape[1];
    int n2 = B->shape[0];
    int p = B->shape[1];
    if (n != n2) {
        fprintf(stderr, "tensor_matmul_cuda_chunked: incompatible internal dimensions (A.cols=%d, B.rows=%d)\n", n, n2);
        exit(EXIT_FAILURE);
    }
    
    size_t num_chunks = (n + chunk_size - 1) / chunk_size;

    size_t bytes_A_sub = m * chunk_size * sizeof(float);
    size_t bytes_B_sub = chunk_size * p * sizeof(float);
    size_t bytes_out_sub = m * p * sizeof(float);

    float *d_A_sub = NULL;
    float *d_B_sub = NULL;
    float *d_out_sub = NULL;

    hipMalloc((void**)&d_A_sub, bytes_A_sub);
    hipMalloc((void**)&d_B_sub, bytes_B_sub);
    hipMalloc((void**)&d_out_sub, bytes_out_sub);

    const int TILE_DIM = 16;
    dim3 threads_per_block(TILE_DIM, TILE_DIM);
    dim3 blocks_per_grid(
        (p + TILE_DIM - 1) / TILE_DIM,
        (m + TILE_DIM - 1) / TILE_DIM
    );

    for (size_t chunk_idx = 0; chunk_idx < num_chunks; ++chunk_idx) {
        int k_start = chunk_idx * chunk_size;
        int n_sub = chunk_size;
        if (k_start + n_sub > n) {
            n_sub = n - k_start;
        }
        
        for (int i = 0; i < m; ++i) {
            const float *host_ptr_A = A->data + (size_t)i * A->stride[0] + (size_t)k_start * A->stride[1];
            
            float *dev_ptr_A = d_A_sub + (size_t)i * n_sub;
            hipMemcpy(dev_ptr_A, host_ptr_A, n_sub * sizeof(float), hipMemcpyHostToDevice);
        }
        
        for (int k_local = 0; k_local < n_sub; ++k_local) {
            const float *host_ptr_B = B->data + (size_t)(k_start + k_local) * B->stride[0];
            float *dev_ptr_B = d_B_sub + (size_t)k_local * p;
            hipMemcpy(dev_ptr_B, host_ptr_B, p * sizeof(float), hipMemcpyHostToDevice);
        }

        hipMemset(d_out_sub, 0, bytes_out_sub);

        matmul_chunk_kernel<<<blocks_per_grid, threads_per_block>>>(d_A_sub, d_B_sub, d_out_sub, m, p, n_sub);
        hipDeviceSynchronize();

        float *h_out_sub = (float*)malloc(bytes_out_sub);
        hipMemcpy(h_out_sub, d_out_sub, bytes_out_sub, hipMemcpyDeviceToHost);

        for (int i = 0; i < m; ++i) {
            for (int j = 0; j < p; ++j) {
                size_t idx_out = (size_t)i * out->stride[0] + (size_t)j * out->stride[1];
                size_t idx_sub = (size_t)i * p + (size_t)j;
                out->data[idx_out] += h_out_sub[idx_sub];
            }
        }
        free(h_out_sub);
    }

    hipFree(d_A_sub);
    hipFree(d_B_sub);
    hipFree(d_out_sub);
}

Tensor* tensor_ones_cuda(int ndim, const int *shape, size_t chunk_size) {
    Tensor *t = tensor_new(ndim, shape);
    if (!t) return NULL;

    size_t N = t->size;
    size_t num_chunks = (N + chunk_size - 1) / chunk_size;
    size_t bytes_chunk = chunk_size * sizeof(float);

    float *d_data;
    hipMalloc((void**)&d_data, bytes_chunk);

    const int threads_per_block = 256;

    for (size_t chunk_idx = 0; chunk_idx < num_chunks; ++chunk_idx) {
        size_t offset = chunk_idx * chunk_size;
        size_t elems_this_chunk = chunk_size;
        if (offset + elems_this_chunk > N) {
            elems_this_chunk = N - offset;
        }
        size_t bytes_this = elems_this_chunk * sizeof(float);

        int blocks_per_grid = (int)((elems_this_chunk + threads_per_block - 1) / threads_per_block);
        tensor_ones_kernel<<<blocks_per_grid, threads_per_block>>>(d_data, elems_this_chunk);
        hipDeviceSynchronize();
        hipMemcpy(t->data + offset, d_data, bytes_this, hipMemcpyDeviceToHost);
    }

    hipFree(d_data);
    return t;
}

Tensor* tensor_zeros_cuda(int ndim, const int *shape, size_t chunk_size) {
    Tensor *t = tensor_new(ndim, shape);
    if (!t) return NULL;

    size_t N = t->size;
    size_t num_chunks = (N + chunk_size - 1) / chunk_size;
    size_t bytes_chunk = chunk_size * sizeof(float);

    float *d_data;
    hipMalloc((void**)&d_data, bytes_chunk);

    const int threads_per_block = 256;

    for (size_t chunk_idx = 0; chunk_idx < num_chunks; ++chunk_idx) {
        size_t offset = chunk_idx * chunk_size;
        size_t elems_this_chunk = chunk_size;
        if (offset + elems_this_chunk > N) {
            elems_this_chunk = N - offset;
        }
        size_t bytes_this = elems_this_chunk * sizeof(float);

        int blocks_per_grid = (int)((elems_this_chunk + threads_per_block - 1) / threads_per_block);
        tensor_zeros_kernel<<<blocks_per_grid, threads_per_block>>>(d_data, elems_this_chunk);
        hipDeviceSynchronize();
        hipMemcpy(t->data + offset, d_data, bytes_this, hipMemcpyDeviceToHost);
    }

    hipFree(d_data);
    return t;
}

Tensor* tensor_rand_cuda(int ndim, const int *shape, size_t chunk_size) {
    Tensor *t = tensor_new(ndim, shape);
    if (!t) return NULL;

    size_t N = t->size;
    size_t num_chunks = (N + chunk_size - 1) / chunk_size;
    size_t bytes_chunk = chunk_size * sizeof(float);

    float *d_data;
    hipMalloc((void**)&d_data, bytes_chunk);

    const int threads_per_block = 256;

    for (size_t chunk_idx = 0; chunk_idx < num_chunks; ++chunk_idx) {
        size_t offset = chunk_idx * chunk_size;
        size_t elems_this_chunk = chunk_size;
        if (offset + elems_this_chunk > N) {
            elems_this_chunk = N - offset;
        }
        size_t bytes_this = elems_this_chunk * sizeof(float);

        int blocks_per_grid = (int)((elems_this_chunk + threads_per_block - 1) / threads_per_block);
        unsigned long long seed = (unsigned long long)time(NULL);
        tensor_rand_kernel<<<blocks_per_grid, threads_per_block>>>(d_data, elems_this_chunk, seed);
        hipDeviceSynchronize();
        hipMemcpy(t->data + offset, d_data, bytes_this, hipMemcpyDeviceToHost);
    }

    hipFree(d_data);
    return t;
}

static size_t tensor_index(const Tensor *t, const int *coords) {
    size_t offset = 0;
    for (int d = 0; d < t->ndim; ++d) {
        if (coords[d] < 0 || coords[d] >= t->shape[d]) {
            fprintf(stderr,
                    "tensor_index: coord %d out of bounds for dimension %d (shape=%d)\n",
                    coords[d], d, t->shape[d]);
            exit(EXIT_FAILURE);
        }
        offset += coords[d] * t->stride[d];
    }
    return offset;
}

static void __tensor_print_recursive(const Tensor *t, int dim, int *coords) {
    if (dim == t->ndim) {
        size_t idx = tensor_index(t, coords);
        printf("%.2f", t->data[idx]);
        return;
    }

    printf("[");
    for (int i = 0; i < t->shape[dim]; ++i) {
        coords[dim] = i;
        __tensor_print_recursive(t, dim + 1, coords);
        if (i < t->shape[dim] - 1) {
            printf(", ");
        }
    }
    printf("]");
}

void tensor_show(Tensor *t) {
    printf("ndim: %d\n", t->ndim);
    printf("size: %zu\n", t->size);

    printf("shape: ");
    __array_print(t->shape, t->ndim, sizeof(int), __int_print);

    printf("stride: ");
    __array_print(t->stride, t->ndim, sizeof(int), __int_print);

    printf("data:\n");
    int *coords = (int*)malloc(t->ndim * sizeof(int));
    if (!coords) {
        fprintf(stderr, "Error allocating coords\n");
        return;
    }
    __tensor_print_recursive(t, 0, coords);
    printf("\n");
    free(coords);
}

void tensor_free(Tensor *t) {
    free(t->shape);
    free(t->stride);
    free(t->data);
    free(t);
}

void cuda_get_info() {
    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("  Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        printf("  Total global memory: %lu\n", prop.totalGlobalMem);
        printf("  Compute capability: %d.%d\n",
               prop.major, prop.minor);
        printf("  Number of SMs: %d\n",
               prop.multiProcessorCount);
        printf("  Max threads per block: %d\n",
               prop.maxThreadsPerBlock);
        printf("  Max threads dimensions: x = %d, y = %d, z = %d\n",
               prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("  Max grid dimensions: x = %d, y = %d, z = %d\n",
               prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    }
}

// int main(void) {
//     int ndim = 2;
//     int shape[ndim] = {6, 6};

//     Tensor *A = tensor_rand_cuda(ndim, shape);
//     tensor_show(A);

//     Tensor *B = tensor_rand_cuda(ndim, shape);
//     tensor_show(B);

//     Tensor *out = tensor_new(ndim, shape);
//     tensor_matmul_cuda(out, A, B);
//     tensor_show(out);

//     return 0;
// }
